// Elapsed Real Time for input-4.txt: 0m1.864s

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Input sequence of integers.
int *vList;

// Number of integers on the list.
int vCount = 0;

// Capacity of the list of integers.
int vCap = 0;

// Target sum.
int target_sum;

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: sequence <target_sum> [report]\n" );
  exit( 1 );
}

// Read the list of values.
__host__ void readList() {
  // Set up initial list and capacity.
  vCap = 5;
  vList = (int *) malloc( vCap * sizeof( int ) );

  // Keep reading as many values as we can.
  int v;
  while ( scanf( "%d\n", &v ) == 1 ) {
    // Grow the list if needed.
    if ( vCount >= vCap ) {
      vCap *= 2;
      vList = (int *) realloc( vList, vCap * sizeof( int ) );
    }

    // Store the latest value in the next array slot.
    vList[ vCount++ ] = v;
  }
}

__global__ void checkSum( int *devList, int *localCounts, int vCount, bool report, int target_sum) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  // citation, this is heavily modified from my previous homeworks (I looked at HW 3 and 4) for this exact problem
  // the approach is similar, but I changed it to be as close as possible to the example output
  // this is the bulk of the code I wrote, so ~ 1/2, but I am pretty sure that is okay.

  // make sure our index is below vCount
  // don't want to go out of bounds
  if ( idx < vCount ) {
    // start our sum at 0
    int sum = 0;
    // start at the end of our section and go backwards checking if we have found our sum
    // something like this if idx = 4 and vCount is 4, we add up the x's values
    // . . . . .
    // . . . . x 
    // . . . x x
    // . . x x x
    // and so on and so forth
    for (int j = idx; j >= 0; j--) {
      // add our x to sum
      sum += devList[j];
      // if sum is target, then increment localCounts
      if (sum == target_sum) {
        localCounts[idx]++; 
        // report if needed
        if (report) { 
          printf("I’m thread %d. Local count: %d Sequence found at: %d-%d.\n", idx, localCounts[idx], j, idx);
        }
      }
    }
    // if localCounts of idx is 0 and we are reporting, then print out special statement
    if (localCounts[idx] == 0 && report) {
      printf("I’m thread %d. Local count: %d\n", idx, 0);
    }
  }
  // end citation
}

int main( int argc, char *argv[] ) {
  if ( argc < 2 || argc > 3 )
    usage();

  if ( sscanf( argv[ 1 ], "%d", &target_sum ) != 1)
    usage();

  bool report = false;
  if ( argc == 3 ) {
    if ( strcmp( argv[ 2 ], "report" ) != 0 )
      usage();
    report = true;
  }
  // should now have vList and vCount
  readList();

  // Add code to allocate memory on the device and copy over the list.
  // citation, got from randomList.cu on moodle. I don't really know if I need to cite this
  // because it is just basic syntax, like if I did printf would I have to cite that?
  int *devList = NULL;
  if ( hipMalloc((void **)&devList, vCount * sizeof(int) ) != hipSuccess )
    fail( "Failed to allocate space for length list on device" );
  // Add code to copy the list over to the device.
  if ( hipMemcpy( devList, vList, vCount * sizeof(int),
                   hipMemcpyHostToDevice) != hipSuccess )
    fail( "Failed to copy list to device" );
  // end citation
  // Add code to allocate space on the device to hold the results.
  int *localCounts = NULL;
  if ( hipMalloc((void **)&localCounts, vCount * sizeof(int) ) != hipSuccess )
    fail( "Failed to allocate space for lenght list on device" );
  // Block and grid dimensions.
  int threadsPerBlock = 1024;
  // Round up for the number of blocks we need.
  int blocksPerGrid = ( vCount + threadsPerBlock - 1 ) / threadsPerBlock;
  printf("Blocks per grid %d\n", blocksPerGrid);
  // Run our kernel on these block/grid dimensions (you'll need to add some parameters)
  checkSum<<<blocksPerGrid, threadsPerBlock>>>( devList, localCounts, vCount, report, target_sum );
  hipError_t error;
  if ( (error = hipGetLastError()) != hipSuccess ) {
    printf("%s\n", hipGetErrorString(error));
    fail( "Failure in CUDA kernel execution." );
  }
  // Add code to copy results back to the host, compare the local maximum lengths
  // and report the final maximum length

  // allocate memory for the counts here
  int * allCounts = (int *) malloc(vCount * sizeof(int));
  // set all counts to 0
  memset(allCounts, 0, vCount * sizeof(int));

  // citation got from randomList.cu from moodle
  if ( hipMemcpy( allCounts, localCounts, vCount * sizeof(int),
                   hipMemcpyDeviceToHost) != hipSuccess )
    fail( "Can't copy list from device to host" );
  // end citation

  int sum = 0;
  for (int i = 0; i < vCount; i++) {
    sum += allCounts[i];
  }
  printf("Total count: %d\n", sum);
  // Free memory on the device and the host.
  // ...
  free( vList );
  free(allCounts);
  hipFree( localCounts );
  hipFree( devList );
  hipDeviceReset();

  return 0;
}
